#include "hip/hip_runtime.h"
#include "cuda_tensor.cuh"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "teachcompute_helpers.h"

namespace cuda_example {

int32_t type_size(hipDataType element_type) {
  switch (element_type) {
  case HIP_R_32I:
  case HIP_R_32F:
    return 4;
  case HIP_R_16F:
  case HIP_R_16BF:
    return 2;
  case HIP_R_8I:
  case HIP_R_8U:
#if defined(CUDA_VERSION) && CUDA_VERSION >= 11080
  case HIP_R_8F_E4M3:
  case HIP_R_8F_E5M2:
#endif
    return 1;
  default:
    NVTE_CHECK(false,
               teachcompute_helpers::MakeString("Unkown data type ", element_type,
                                                 " and this CUDA version ", CUDA_VERSION, "."));
  }
}

void TensorData::allocate(hipDataType dtype, std::size_t size, TensorDevice device) {
  this->dtype = dtype;
  this->size = size;
  this->device = device;
  switch (device) {
  case TensorDevice::CPU:
    dptr = malloc(size * type_size(dtype));
    break;
  case TensorDevice::CUDA:
    if (hipMalloc(&dptr, size * type_size(dtype)) != hipSuccess) {
      NVTE_ERROR(
          teachcompute_helpers::MakeString("Unable to allocate ", size, " bytes on GPU."));
    }
    break;
  }
}

void TensorData::free() {
  if (dptr != nullptr) {
    switch (device) {
    case TensorDevice::CPU:
      ::free(dptr);
      break;
    case TensorDevice::CUDA:
      NVTE_CHECK_CUDA(hipFree(dptr));
      break;
    }
    dptr = nullptr;
  }
}

void TensorData::copy_from_cpu(void *ptr) {
  switch (device) {
  case TensorDevice::CPU:
    memcpy(dptr, ptr, type_size(dtype) * size);
    break;
  case TensorDevice::CUDA:
    NVTE_CHECK_CUDA(hipMemcpy(dptr, ptr, type_size(dtype) * size, hipMemcpyHostToDevice));
    break;
  default:
    NVTE_CHECK(false, teachcompute_helpers::MakeString("Unsupported device ", (int)device,
                                                        " for copy_from_cpu."));
  }
}

Tensor::Tensor(const char *name, std::size_t size, hipDataType dtype, TensorDevice device,
               TensorDevice scale_device) {
  this->name = name;
  data.allocate(dtype, size, device);
}

Tensor::~Tensor() {
  data.free();
  scale.free();
  scale_inv.free();
  amax.free();
}

__global__ void generateRandomFloat16(__half *randomFloat16, int numElements,
                                      unsigned int seed) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < numElements) {
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);
    float randValue = hiprand_uniform(&state);
    randomFloat16[tid] = __float2half(randValue);
  }
}

__global__ void generateRandomBFloat16(__hip_bfloat16 *randomFloat16, int numElements,
                                       unsigned int seed) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < numElements) {
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);
    float randValue = hiprand_uniform(&state);
    randomFloat16[tid] = __float2bfloat16(randValue);
  }
}

__global__ void generateRandomInt8x4(int *randomInt8, int numElements, unsigned int seed) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < numElements / 4) {
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);
    int randValue = hiprand_poisson(&state, 1);
    randomInt8[tid] = randValue;
  }
}

void Tensor::rnd() {
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  switch (data.dtype) {
  case HIP_R_32F:
    hiprandGenerateUniform(gen, static_cast<float *>(data.dptr), data.size);
    break;
  case HIP_R_16F: {
    int blockSize = 256;
    int numBlocks = (data.size + blockSize - 1) / blockSize;
    generateRandomFloat16<<<numBlocks, blockSize>>>(static_cast<__half *>(data.dptr), data.size,
                                                    0);
    hipDeviceSynchronize();
  } break;
  case HIP_R_16BF: {
    int blockSize = 256;
    int numBlocks = (data.size + blockSize - 1) / blockSize;
    generateRandomBFloat16<<<numBlocks, blockSize>>>(static_cast<__hip_bfloat16 *>(data.dptr),
                                                     data.size, 0);
    hipDeviceSynchronize();
  } break;
  case HIP_R_8I: {
    int blockSize = 256;
    int numBlocks = (data.size + blockSize - 1) / blockSize;
    generateRandomInt8x4<<<numBlocks, blockSize>>>(static_cast<int *>(data.dptr), data.size, 0);
    hipDeviceSynchronize();
  } break;
  default:
    NVTE_CHECK(false, teachcompute_helpers::MakeString("Unsupported dtype ", data.dtype,
                                                        " for rnd."));
  }
  hiprandDestroyGenerator(gen);
}

} // namespace cuda_example